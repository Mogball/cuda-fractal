#include "hip/hip_runtime.h"
#include <stdint.h>
#include <gpu.h>
#include <gpu_error.cuh>

// Hardcoded range on Mandelbrot set, which ranges
// from X: (-2, 1) and Y: (-1, 1)
#define MIN_X   (-2.0)
#define MAX_X   (+1.0)
#define MIN_Y   (-1.0)
#define MAX_Y   (+1.0)
#define RANGE_X (MAX_X - MIN_X)
#define RANGE_Y (MAX_Y - MIN_Y)

/**
 * Device function to compute ARGB pixel.
 * @param r 8-bit red value
 * @param g 8-bit green value
 * @param b 8-bit blue value
 * @return ARGB pixel with max alpha
 */
__device__
uint32_t argb(uint8_t r, uint8_t g, uint8_t b) {
    return (0xff << 24) | (r << 16) | (g << 8) | b;
}

/**
 * Basic coloring with greyscale.
 * @param t escape time for the pixel
 * @param T max iterations
 * @return greyscale ARGB pixel
 */
__device__
uint32_t color_greyscale(long t, long T) {
    uint8_t grey = (uint8_t) (t / (double) T * 0xff);
    return argb(grey, grey, grey);
}

/**
 * Device function to compute Mandelbrot escape time for
 * a particular pixel, or coordinate (x0, y0).
 * @param x0 x-value
 * @param y0 y-value
 * @param T  max iterations
 * @return Mandelbrot escape time
 */
__device__
long mandelbrot_compute(double x0, double y0, long T) {
    double x = 0;
    double y = 0;
    double xSq = 0;
    double ySq = 0;
    long t = 0;
    while (xSq + ySq < 4 && t < T) {
        y = x * y;
        y += y;
        y += y0;
        x = xSq - ySq + x0;
        xSq = x * x;
        ySq = y * y;
        ++t;
    }
    return t;
}

/**
 * Kernel to compute Mandelbrot value at pixels.
 * @param config fractal rendering config
 * @param data   array to store pixel data
 */
__global__
void mandelbrot(render_config config, uint32_t *data) {
    int index_x = blockIdx.x * blockDim.x + threadIdx.x;
    int index_y = blockIdx.y * blockDim.y + threadIdx.y;
    int stride_x = gridDim.x * blockDim.x;
    int stride_y = gridDim.y * blockDim.y;
    double m = (double) config.max;
    double x0, y0;
    long t;
    for (int i_x = index_x; i_x < m; i_x += stride_x) {
        for (int i_y = index_y; i_y < m; i_y += stride_y) {
            x0 = (i_x / m - 0.5) * config.s + config.xc;
            y0 = (i_y / m - 0.5) * config.s + config.yc;
            t = mandelbrot_compute(x0, y0, config.T);
            *(data + i_x + i_y * config.max) = color_greyscale(t, config.T);
        }
    }
}

/**
 * Exposed function to used GPU to compute Mandelbrot data.
 * @param config fractal rendering configuration
 * @param data   array to store pixel data
 */
void gpu_mandelbrot(const render_config &config, uint32_t *data) {
    // Allocate device image data
    uint32_t *device_data;
    uint32_t num_pixels = config.max * config.max;
    uint32_t num_bytes = num_pixels * sizeof(uint32_t);
    errchk( hipMalloc(&device_data, num_bytes) );

    // Launch Mandelbrot kernels
    dim3 block_dim(16, 16);
    dim3 block_num(
        (config.max + block_dim.x - 1) / block_dim.x,
        (config.max + block_dim.y - 1) / block_dim.y
    );
    mandelbrot<<<block_num, block_dim>>>(config, device_data);
    errchk( hipPeekAtLastError()   );
    errchk( hipDeviceSynchronize() );

    // Copy data into host memory
    errchk( hipMemcpy(data, device_data, num_bytes, hipMemcpyDeviceToHost) );
    errchk( hipFree(device_data) );
}
